#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/sort.h>
using namespace std;
/**********************************************************
***********************************************************
error checking stufff
***********************************************************
***********************************************************/

// Enable this for error checking
#define CUDA_CHECK_ERROR
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )



inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
	#ifdef CUDA_CHECK_ERROR
	#pragma warning( push )
	#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{
		if ( hipSuccess != err )
		{
			fprintf( stderr,"cudaSafeCall() failed at %s:%i : %s\n",file, line, hipGetErrorString( err ) );
			exit( -1 );
		}
	} while ( 0 );
	#pragma warning( pop )
	#endif // CUDA_CHECK_ERROR
	return;
}


inline void __cudaCheckError( const char *file, const int line )
{
	#ifdef CUDA_CHECK_ERROR
	#pragma warning( push )
	#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{
		hipError_t err = hipGetLastError();
		if ( hipSuccess != err )
		{
			fprintf( stderr,"cudaCheckError() failed at %s:%i : %s.\n",file, line, hipGetErrorString( err ) );
			exit( -1 );
		}
		// More careful checking. However, this will affect performance.
		// Comment if not needed.
		err = hipDeviceSynchronize();
		if( hipSuccess != err )
		{
			fprintf( stderr,"cudaCheckError() with sync failed at %s:%i : %s.\n",file, line, hipGetErrorString( err ) );
			exit( -1 );
		}
	} while ( 0 );
	#pragma warning( pop )
	#endif // CUDA_CHECK_ERROR
	return;
}




/***************************************************************
***************************************************************
end of error checking stuff
****************************************************************
***************************************************************/





// function takes an array pointer, and the number of rows and cols in the array, and
// allocates and intializes the array to a bunch of random numbers
// Note that this function creates a 1D array that is a flattened 2D array
// to access data item data[i][j], you must can use data[(i*rows) + j]
int * makeRandArray( const int size, const int seed )
{
	srand( seed );
	int * array = new int[ size ];
	for( int i = 0; i < size; i ++ ) {
		array[i] = std::rand() % 1000000;
	}
	return array;
}




//*******************************//
// In place radix sort           //
//*******************************//
__global__ void matavgKernel(int* array, const int S, int *out)
{

	const int size = S;
	int max = array[0];

	//find largest item in array so we know how many digits there are
	for(int i = 1; i < size; i++)
	{
		if(array[i] > max)
			max = array[i];
	}


	//go through each digit
	for(int exp = 1; max/exp > 0; exp *= 10){
		int *output = (int *)malloc(sizeof(int)*size);

		int i = 0;
		
		//array of size ten, for each possible value at this digit		
		int *count = (int *)malloc(sizeof(int)*10);

		//get the count of how many items have each possible value at this digit
		//(ie number of "2"s, "3"s, etc.)
		//this is the part that will be parallelized for the multithreading version
		for(i = 0; i < size; i++){
			count[ (array[i]/exp)%10 ]++;
		}

		//aggregate previous values into count, getting a count of values that size and smaller at digit exp.
		for(i = 1; i < 10; i++){
			count[i] += count[i - 1];
		}

		//based off known number of items in each location, copy things into output sorted by index exp
		for(i = size-1; i >= 0; i--){
			output[count[ (array[i]/exp)%10 ] - 1] = array[i];
			count[ (array[i]/exp)%10 ]--;
		}

		//copy back into original array so ordering of this digit is preserved for future iterations
		for(i = 0; i < size; i++){
			array[i] = output[i];
			
		}
	}
}






int main( int argc, char * argv[] )
{
	int * array; // the poitner to the array of rands
	int size, seed; // values for the size of the array
	bool printSorted = false;
	// and the seed for generating
	// random numbers
	// check the command line args
	if( argc < 4 ){
		std::cerr << "usage: " << argv[0] << " [amount of random nums to generate] [seed value for rand]" << " [1 to print sorted array, 0 otherwise]" << std::endl;
		exit( -1 );
	}


	// convert cstrings to ints
	{
		std::stringstream ss1( argv[1] );
		ss1 >> size;
	}

	{
		std::stringstream ss1( argv[2] );
		ss1 >> seed;
	}

	{
		int sortPrint;
		std::stringstream ss1( argv[3] );
		ss1 >> sortPrint;
		if( sortPrint == 1 )
			printSorted = true;
	}

	// get the random numbers
	array = makeRandArray( size, seed );


	//print out initial array
	//for(int index = 0; index < size-1; index++)
	//{
	//	printf("%d, ", array[index]);
	//}
	//printf("%d\n", array[size-1]);

	/***********************************
	create a cuda timer to time execution
	*
	**********************************/

	hipEvent_t startTotal, stopTotal;
	float timeTotal;
	hipEventCreate(&startTotal);
	hipEventCreate(&stopTotal);
	hipEventRecord( startTotal, 0 );

	/***********************************
	end of cuda timer creation
	*
	**********************************/



	/////////////////////////////////////////////////////////////////////
	///////////////////////    YOUR CODE HERE     ///////////////////////
	/////////////////////////////////////////////////////////////////////

	//STEP 1
	int *device_nums;
	hipMalloc( &device_nums, size * sizeof(int) );	//this will be the array on the GPU

	//STEP 2
	// copy our array into our array on GPU
	hipMemcpy(device_nums, &(array[0]), size * sizeof(int), hipMemcpyHostToDevice);


	//STEP 3
	int* device_count;
	hipMalloc( &device_count, sizeof(int) );
	{
	  int startCount = 0;
	  hipMemcpy(device_count, &(startCount), sizeof(int), hipMemcpyHostToDevice);
	}


	//STEP 4
	//dim3 threadsPerBlock(1, 1);
	//dim3 numBlocks((size +threadsPerBlock.x-1) / threadsPerBlock.x, (size +threadsPerBlock.y-1) / threadsPerBlock.y);


	//STEP 5
	matavgKernel<<< 1, 1>>> ( device_nums, size, device_count );


	int totalFound;
	hipMemcpy( array, device_nums, size*sizeof(int), hipMemcpyDeviceToHost );

	






	/*
	You need to implement your kernel as a function at the top of this file.
	Here you must
	1) allocate device memory
	2) set up the grid and block sizes
	3) call your kenrnel
	4) get the result back from the GPU
	to use the error checking code, wrap any cudamalloc functions as follows:
	CudaSafeCall( hipMalloc( &pointer_to_a_device_pointer,
	length_of_array * sizeof( int ) ) );
	Also, place the following function call immediately after you call your kernel
	( or after any other cuda call that you think might be causing an error )
	CudaCheckError();
	*/



	/***********************************
	Stop and destroy the cuda timer
	*
	**********************************/

	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );

	/***********************************
	end of cuda timer destruction
	*
	**********************************/



	std::cerr << "Total time in seconds: "	<< timeTotal / 1000.0 << std::endl;


	if( printSorted ){
		//print out sorted array
		for(int index = 0; index < size-1; index++)
		{
			printf("%d, ", array[index]);
	        }
		printf("%d\n", array[size-1]);

	}
}
